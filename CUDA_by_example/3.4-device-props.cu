#include "book.h"

int main(void) {
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR( hipGetDevice(&dev));
    printf("ID current CUDA device: %d\n", dev);

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 8;
    prop.minor = 9;
    HANDLE_ERROR( hipChooseDevice(&dev, &prop));
    printf( "ID of CUDA device closest to revision 8.9: %d\n", dev );
    HANDLE_ERROR( hipSetDevice( dev ) );

}